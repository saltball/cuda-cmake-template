/*
*hello_world.cu
*/

#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void hello_world(void)
{
  printf("GPU: Hello world from thread %d of total grid %d, block %d of total blocks %d\n", threadIdx.x, gridDim.x, blockIdx.x, blockDim.x);
}
int main(int argc,char **argv)
{
  printf("CPU: Hello world!\n");
  hello_world<<<2,10>>>();
  hipDeviceReset();//if no this line ,it can not output hello world from gpu
  return 0;
}