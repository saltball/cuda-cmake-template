#include "hip/hip_runtime.h"
#include "common.h"
#include <time.h>
#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif
#ifdef _WIN32
int gettimeofday(struct timeval *tp, void *tzp)
{
  time_t clock;
  struct tm tm;
  SYSTEMTIME wtm;
  GetLocalTime(&wtm);
  tm.tm_year = wtm.wYear - 1900;
  tm.tm_mon = wtm.wMonth - 1;
  tm.tm_mday = wtm.wDay;
  tm.tm_hour = wtm.wHour;
  tm.tm_min = wtm.wMinute;
  tm.tm_sec = wtm.wSecond;
  tm.tm_isdst = -1;
  clock = mktime(&tm);
  tp->tv_sec = clock;
  tp->tv_usec = wtm.wMilliseconds * 1000;
  return (0);
}
#endif
double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}
void initialData(float *ip, int size)
{
  time_t t;
  srand((unsigned)time(&t));
  for (int i = 0; i < size; i++)
  {
    ip[i] = (float)(rand() & 0xffff) / 1000.0f;
  }
}
void initialData_int(int *ip, int size)
{
  time_t t;
  srand((unsigned)time(&t));
  for (int i = 0; i < size; i++)
  {
    ip[i] = int(rand() & 0xff);
  }
}
void printMatrix(float *C, const int nx, const int ny)
{
  float *ic = C;
  std::cout<<fmt::format("Matrix<{},{}>:", ny, nx);
  for (int i = 0; i < ny; i++)
  {
    for (int j = 0; j < nx; j++)
    {
      std::cout<<fmt::format("{:6f} ", C[j]);
    }
    ic += nx;
    std::cout<<fmt::format("\n");
  }
}
#ifdef __HIPCC__
void initDevice(int devNum)
{
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  std::cout<<fmt::format("Using device {}: {}\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));
}
void checkResult(float *hostRef, float *gpuRef, const int N)
{
  double epsilon = 1.0E-8;
  for (int i = 0; i < N; i++)
  {
    if (fabs(hostRef[i] - gpuRef[i]) > epsilon)
    {
      std::cout<<fmt::format("Results don\'t match!\n");
      std::cout<<fmt::format("{}(hostRef[{}] )!= {}(gpuRef[{}])\n", hostRef[i], i, gpuRef[i], i);
      return;
    }
  }
  std::cout<<fmt::format("Check result success!\n");
}
#endif