#include "hip/hip_runtime.h"
#include "common.cpp"
#ifdef __HIPCC__
void initDevice(int devNum)
{
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  std::cout<<fmt::format("Using device {}: {}\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));
}
void checkResult(float *hostRef, float *gpuRef, const int N)
{
  double epsilon = 1.0E-8;
  for (int i = 0; i < N; i++)
  {
    if (fabs(hostRef[i] - gpuRef[i]) > epsilon)
    {
      std::cout<<fmt::format("Results don\'t match!\n");
      std::cout<<fmt::format("{}(hostRef[{}] )!= {}(gpuRef[{}])\n", hostRef[i], i, gpuRef[i], i);
      return;
    }
  }
  std::cout<<fmt::format("Check result success!\n");
}
#endif