#include "hip/hip_runtime.h"
#include "op.h"
__global__ void hello_world(void)
{
  printf("GPU: Hello world from thread %d of total grid %d, block %d of total blocks %d. As %d th item.\n", threadIdx.x, gridDim.x, blockIdx.x, blockDim.x, blockIdx.x * blockDim.x + threadIdx.x);
}
int hello()
{
  printf("CPU: Hello world!\n");
  initDevice(0);
  double start_t = cpuSecond();
  hello_world<<<11, 7>>>();
  hipDeviceReset(); // if no this line ,it can not output hello world from gpu
  double end_t = cpuSecond();
  printf("CPU: Time taken: %f\n", end_t - start_t);
  return 0;
}