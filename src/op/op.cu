#include "hip/hip_runtime.h"
#include "op.h"
#include <stdio.h>
__global__ void hello_world(void)
{
  printf("GPU: Hello world from thread %d of total grid %d, block %d of total blocks %d. As %d th item.\n", threadIdx.x, gridDim.x, blockIdx.x, blockDim.x, blockIdx.x * blockDim.x + threadIdx.x);
}
int hello()
{
  std::cout<<fmt::format("CPU: Hello world!\n");
  initDevice(0);
  double start_t = cpuSecond();
  hello_world<<<11, 7>>>();
  hipDeviceReset(); // if no this line ,it can not output hello world from gpu
  double end_t = cpuSecond();
  std::cout<<fmt::format("CPU: Time taken: {:.6f}\n", end_t - start_t);
  return 0;
}