#include "hip/hip_runtime.h"
#include "op.h"
#include <hipsolver.h>

__global__ void op_kernal(double *a, double *b, double *c)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;
  int idx = i + blockDim.x * j + blockDim.x * blockDim.y * k;
  // printf("GPU: threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d) gridDim(%d,%d,%d)\n. GPU: idx = % d,  a[i] = % d\n",
  //        threadIdx.x, threadIdx.y, threadIdx.z,
  //        blockIdx.x, blockIdx.y, blockIdx.z,
  //        blockDim.x, blockDim.y, blockDim.z,
  //        gridDim.x, gridDim.y, gridDim.z,
  //        idx,
  //        a[idx]);
  c[idx] = a[idx] + b[idx];
}
int op(int dim_x, int dim_y, int dim_z)
{
  std::cout << fmt::format("CPU: Hello world!\n");
  int size_dim = dim_x * dim_y * dim_z;
  std::cout << fmt::format("CPU: size_dim = {}\n", size_dim);
  initDevice(0);

  double *a_h = (double *)malloc(size_dim * sizeof(double));
  double *b_h = (double *)malloc(size_dim * sizeof(double));
  double *c_h = (double *)malloc(size_dim * sizeof(double));
  double *c_from_gpu_h = (double *)malloc(size_dim * sizeof(double));

  memset(c_h, 0, size_dim * sizeof(double));

  double *a_d, *b_d, *c_d;
  CHECK(hipMalloc((double **)&a_d, size_dim * sizeof(double)));
  CHECK(hipMalloc((double **)&b_d, size_dim * sizeof(double)));
  CHECK(hipMalloc((double **)&c_d, size_dim * sizeof(double)));

  initialData(a_h, size_dim);
  initialData(b_h, size_dim);

  CHECK(hipMemcpy(a_d, a_h, size_dim * sizeof(double), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d, b_h, size_dim * sizeof(double), hipMemcpyHostToDevice));

  dim3 block(dim_x,dim_y*dim_z,1);
  dim3 grid(1, 1, 1);

  double start_t = cpuSecond();
  op_kernal<<<grid, block>>>(a_d, b_d, c_d);
  op_kernal<<<grid, block>>>(b_d, b_d, c_d);
  op_kernal<<<grid, block>>>(a_d, a_d, c_d);

  CHECK(hipMemcpy(c_from_gpu_h, c_d, size_dim * sizeof(double), hipMemcpyDeviceToHost));

  hipDeviceReset(); // if no this line ,it can not output hello world from gpu

  // printVector(c_from_gpu_h, dim);
  double end_t = cpuSecond();
  std::cout << fmt::format("CPU: Time taken: {:.6f}\n", end_t - start_t);
  free(a_h);
  free(b_h);
  free(c_h);
  free(c_from_gpu_h);
  return 0;
}